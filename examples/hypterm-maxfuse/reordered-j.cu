#include <stdio.h>
#include "hip/hip_runtime.h"
#define max(x,y)  ((x) > (y)? (x) : (y))
#define min(x,y)  ((x) < (y)? (x) : (y))
#define ceil(a,b) ((a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1)

void check_error (const char* message) {
	hipError_t error = hipGetLastError ();
	if (error != hipSuccess) {
		printf ("CUDA error : %s, %s\n", message, hipGetErrorString (error));
		exit(-1);
	}
}

__global__ void hypterm (double * __restrict__ flux_in_0, double * __restrict__ flux_in_1, double * __restrict__ flux_in_2, double * __restrict__ flux_in_3, double * __restrict__ flux_in_4, double * __restrict__ cons_in_1, double * __restrict__ cons_in_2, double * __restrict__ cons_in_3, double * __restrict__ cons_in_4, double * __restrict__ q_in_1, double * __restrict__ q_in_2, double * __restrict__ q_in_3, double * __restrict__ q_in_4, double dxinv0, double dxinv1, double dxinv2, int L, int M, int N) {
	//Determing the block's indices
	int blockdim_i= (int)(blockDim.x);
	int i0 = (int)(blockIdx.x)*(blockdim_i);
	int i = max (i0, 0) + (int)(threadIdx.x);
	int blockdim_j= (int)(blockDim.y);
	int j0 = (int)(blockIdx.y)*(blockdim_j);
	int j = max (j0, 0) + (int)(threadIdx.y);
	int blockdim_k= (int)(blockDim.z);
	int k0 = (int)(blockIdx.z)*(blockdim_k);
	int k = max (k0, 0) + (int)(threadIdx.z);

	double (*flux_0)[308][308] = (double (*)[308][308])flux_in_0;
	double (*flux_1)[308][308] = (double (*)[308][308])flux_in_1;
	double (*flux_2)[308][308] = (double (*)[308][308])flux_in_2;
	double (*flux_3)[308][308] = (double (*)[308][308])flux_in_3;
	double (*flux_4)[308][308] = (double (*)[308][308])flux_in_4;
	double (*q_1)[308][308] = (double (*)[308][308])q_in_1;
	double (*q_2)[308][308] = (double (*)[308][308])q_in_2;
	double (*q_3)[308][308] = (double (*)[308][308])q_in_3;
	double (*q_4)[308][308] = (double (*)[308][308])q_in_4;
	double (*cons_1)[308][308] = (double (*)[308][308])cons_in_1;
	double (*cons_2)[308][308] = (double (*)[308][308])cons_in_2;
	double (*cons_3)[308][308] = (double (*)[308][308])cons_in_3;
	double (*cons_4)[308][308] = (double (*)[308][308])cons_in_4;

	if (i>=4 & j>=4 & k>=4 & i<=N-5 & j<=N-5 & k<=N-5) {
double _t_1_;
double _t_0_;
double _t_2_;
double _t_3_;
double _t_4_;
double flux_0kc0jc0ic0;
double _t_6_;
double _t_5_;
double _t_7_;
double _t_8_;
double _t_9_;
double flux_1kc0jc0ic0;
double _t_11_;
double _t_10_;
double _t_12_;
double _t_13_;
double _t_14_;
double flux_2kc0jc0ic0;
double _t_16_;
double _t_15_;
double _t_17_;
double _t_18_;
double _t_19_;
double flux_3kc0jc0ic0;
double _t_21_;
double _t_20_;
double _t_22_;
double _t_23_;
double _t_24_;
double flux_4kc0jc0ic0;
double _t_27_;
double _t_26_;
double _t_28_;
double _t_29_;
double _t_30_;
double _t_25_;
double _t_33_;
double _t_32_;
double _t_34_;
double _t_35_;
double _t_36_;
double _t_31_;
double _t_39_;
double _t_38_;
double _t_40_;
double _t_41_;
double _t_42_;
double _t_37_;
double _t_45_;
double _t_44_;
double _t_46_;
double _t_47_;
double _t_48_;
double _t_43_;
double _t_51_;
double _t_50_;
double _t_52_;
double _t_53_;
double _t_54_;
double _t_49_;

_t_1_ = cons_1[k][j][i+1];
_t_1_ -= cons_1[k][j][i-1];
_t_0_ = 0.8 * _t_1_;
_t_2_ = cons_1[k][j][i+2];
_t_2_ -= cons_1[k][j][i-2];
_t_0_ -= 0.2 * _t_2_;
_t_3_ = cons_1[k][j][i+3];
_t_3_ -= cons_1[k][j][i-3];
_t_0_ += 0.038 * _t_3_;
_t_4_ = cons_1[k][j][i+4];
_t_4_ -= cons_1[k][j][i-4];
_t_0_ -= 0.0035 * _t_4_;
flux_0kc0jc0ic0 = _t_0_ * dxinv0;
_t_6_ = cons_1[k][j][i+1] * q_1[k][j][i+1];
_t_6_ -= cons_1[k][j][i-1] * q_1[k][j][i-1];
_t_6_ += q_4[k][j][i+1];
_t_6_ -= q_4[k][j][i-1];
_t_5_ = 0.8 * _t_6_;
_t_7_ = cons_1[k][j][i+2] * q_1[k][j][i+2];
_t_7_ -= cons_1[k][j][i-2] * q_1[k][j][i-2];
_t_7_ += q_4[k][j][i+2];
_t_7_ -= q_4[k][j][i-2];
_t_5_ -= 0.2 * _t_7_;
_t_8_ = cons_1[k][j][i+3] * q_1[k][j][i+3];
_t_8_ -= cons_1[k][j][i-3] * q_1[k][j][i-3];
_t_8_ += q_4[k][j][i+3];
_t_8_ -= q_4[k][j][i-3];
_t_5_ += 0.038 * _t_8_;
_t_9_ = cons_1[k][j][i+4] * q_1[k][j][i+4];
_t_9_ -= cons_1[k][j][i-4] * q_1[k][j][i-4];
_t_9_ += q_4[k][j][i+4];
_t_9_ -= q_4[k][j][i-4];
_t_5_ -= 0.0035 * _t_9_;
flux_1kc0jc0ic0 = _t_5_ * dxinv0;
_t_11_ = cons_2[k][j][i+1] * q_1[k][j][i+1];
_t_11_ -= cons_2[k][j][i-1] * q_1[k][j][i-1];
_t_10_ = 0.8 * _t_11_;
_t_12_ = cons_2[k][j][i+2] * q_1[k][j][i+2];
_t_12_ -= cons_2[k][j][i-2] * q_1[k][j][i-2];
_t_10_ -= 0.2 * _t_12_;
_t_13_ = cons_2[k][j][i+3] * q_1[k][j][i+3];
_t_13_ -= cons_2[k][j][i-3] * q_1[k][j][i-3];
_t_10_ += 0.038 * _t_13_;
_t_14_ = cons_2[k][j][i+4] * q_1[k][j][i+4];
_t_14_ -= cons_2[k][j][i-4] * q_1[k][j][i-4];
_t_10_ -= 0.0035 * _t_14_;
flux_2kc0jc0ic0 = _t_10_ * dxinv0;
_t_16_ = cons_3[k][j][i+1] * q_1[k][j][i+1];
_t_16_ -= cons_3[k][j][i-1] * q_1[k][j][i-1];
_t_15_ = 0.8 * _t_16_;
_t_17_ = cons_3[k][j][i+2] * q_1[k][j][i+2];
_t_17_ -= cons_3[k][j][i-2] * q_1[k][j][i-2];
_t_15_ -= 0.2 * _t_17_;
_t_18_ = cons_3[k][j][i+3] * q_1[k][j][i+3];
_t_18_ -= cons_3[k][j][i-3] * q_1[k][j][i-3];
_t_15_ += 0.038 * _t_18_;
_t_19_ = cons_3[k][j][i+4] * q_1[k][j][i+4];
_t_19_ -= cons_3[k][j][i-4] * q_1[k][j][i-4];
_t_15_ -= 0.0035 * _t_19_;
flux_3kc0jc0ic0 = _t_15_ * dxinv0;
_t_21_ = cons_4[k][j][i+1] * q_1[k][j][i+1];
_t_21_ -= cons_4[k][j][i-1] * q_1[k][j][i-1];
_t_21_ += q_4[k][j][i+1] * q_1[k][j][i+1];
_t_21_ -= q_4[k][j][i-1] * q_1[k][j][i-1];
_t_20_ = 0.8 * _t_21_;
_t_22_ = cons_4[k][j][i+2] * q_1[k][j][i+2];
_t_22_ -= cons_4[k][j][i-2] * q_1[k][j][i-2];
_t_22_ += q_4[k][j][i+2] * q_1[k][j][i+2];
_t_22_ -= q_4[k][j][i-2] * q_1[k][j][i-2];
_t_20_ -= 0.2 * _t_22_;
_t_23_ = cons_4[k][j][i+3] * q_1[k][j][i+3];
_t_23_ -= cons_4[k][j][i-3] * q_1[k][j][i-3];
_t_23_ += q_4[k][j][i+3] * q_1[k][j][i+3];
_t_23_ -= q_4[k][j][i-3] * q_1[k][j][i-3];
_t_20_ += 0.038 * _t_23_;
_t_24_ = cons_4[k][j][i+4] * q_1[k][j][i+4];
_t_24_ -= cons_4[k][j][i-4] * q_1[k][j][i-4];
_t_24_ += q_4[k][j][i+4] * q_1[k][j][i+4];
_t_24_ -= q_4[k][j][i-4] * q_1[k][j][i-4];
_t_20_ -= 0.0035 * _t_24_;
flux_4kc0jc0ic0 = _t_20_ * dxinv0;
_t_27_ = cons_2[k][j+1][i];
_t_27_ -= cons_2[k][j-1][i];
_t_26_ = 0.8 * _t_27_;
_t_28_ = cons_2[k][j+2][i];
_t_28_ -= cons_2[k][j-2][i];
_t_26_ -= 0.2 * _t_28_;
_t_29_ = cons_2[k][j+3][i];
_t_29_ -= cons_2[k][j-3][i];
_t_26_ += 0.038 * _t_29_;
_t_30_ = cons_2[k][j+4][i];
_t_30_ -= cons_2[k][j-4][i];
_t_26_ -= 0.0035 * _t_30_;
_t_25_ = _t_26_ * dxinv1;
flux_0kc0jc0ic0 -= _t_25_;
flux_0[k][j][i] = flux_0kc0jc0ic0;
_t_33_ = cons_1[k][j+1][i] * q_2[k][j+1][i];
_t_33_ -= cons_1[k][j-1][i] * q_2[k][j-1][i];
_t_32_ = 0.8 * _t_33_;
_t_34_ = cons_1[k][j+2][i] * q_2[k][j+2][i];
_t_34_ -= cons_1[k][j-2][i] * q_2[k][j-2][i];
_t_32_ -= 0.2 * _t_34_;
_t_35_ = cons_1[k][j+3][i] * q_2[k][j+3][i];
_t_35_ -= cons_1[k][j-3][i] * q_2[k][j-3][i];
_t_32_ += 0.038 * _t_35_;
_t_36_ = cons_1[k][j+4][i] * q_2[k][j+4][i];
_t_36_ -= cons_1[k][j-4][i] * q_2[k][j-4][i];
_t_32_ -= 0.0035 * _t_36_;
_t_31_ = _t_32_ * dxinv1;
flux_1kc0jc0ic0 -= _t_31_;
flux_1[k][j][i] = flux_1kc0jc0ic0;
_t_39_ = cons_2[k][j+1][i] * q_2[k][j+1][i];
_t_39_ -= cons_2[k][j-1][i] * q_2[k][j-1][i];
_t_39_ += q_4[k][j+1][i];
_t_39_ -= q_4[k][j-1][i];
_t_38_ = 0.8 * _t_39_;
_t_40_ = cons_2[k][j+2][i] * q_2[k][j+2][i];
_t_40_ -= cons_2[k][j-2][i] * q_2[k][j-2][i];
_t_40_ += q_4[k][j+2][i];
_t_40_ -= q_4[k][j-2][i];
_t_38_ -= 0.2 * _t_40_;
_t_41_ = cons_2[k][j+3][i] * q_2[k][j+3][i];
_t_41_ -= cons_2[k][j-3][i] * q_2[k][j-3][i];
_t_41_ += q_4[k][j+3][i];
_t_41_ -= q_4[k][j-3][i];
_t_38_ += 0.038 * _t_41_;
_t_42_ = cons_2[k][j+4][i] * q_2[k][j+4][i];
_t_42_ -= cons_2[k][j-4][i] * q_2[k][j-4][i];
_t_42_ += q_4[k][j+4][i];
_t_42_ -= q_4[k][j-4][i];
_t_38_ -= 0.0035 * _t_42_;
_t_37_ = _t_38_ * dxinv1;
flux_2kc0jc0ic0 -= _t_37_;
flux_2[k][j][i] = flux_2kc0jc0ic0;
_t_45_ = cons_3[k][j+1][i] * q_2[k][j+1][i];
_t_45_ -= cons_3[k][j-1][i] * q_2[k][j-1][i];
_t_44_ = 0.8 * _t_45_;
_t_46_ = cons_3[k][j+2][i] * q_2[k][j+2][i];
_t_46_ -= cons_3[k][j-2][i] * q_2[k][j-2][i];
_t_44_ -= 0.2 * _t_46_;
_t_47_ = cons_3[k][j+3][i] * q_2[k][j+3][i];
_t_47_ -= cons_3[k][j-3][i] * q_2[k][j-3][i];
_t_44_ += 0.038 * _t_47_;
_t_48_ = cons_3[k][j+4][i] * q_2[k][j+4][i];
_t_48_ -= cons_3[k][j-4][i] * q_2[k][j-4][i];
_t_44_ -= 0.0035 * _t_48_;
_t_43_ = _t_44_ * dxinv1;
flux_3kc0jc0ic0 -= _t_43_;
flux_3[k][j][i] = flux_3kc0jc0ic0;
_t_51_ = cons_4[k][j+1][i] * q_2[k][j+1][i];
_t_51_ -= cons_4[k][j-1][i] * q_2[k][j-1][i];
_t_51_ += q_4[k][j+1][i] * q_2[k][j+1][i];
_t_51_ -= q_4[k][j-1][i] * q_2[k][j-1][i];
_t_50_ = 0.8 * _t_51_;
_t_52_ = cons_4[k][j+2][i] * q_2[k][j+2][i];
_t_52_ -= cons_4[k][j-2][i] * q_2[k][j-2][i];
_t_52_ += q_4[k][j+2][i] * q_2[k][j+2][i];
_t_52_ -= q_4[k][j-2][i] * q_2[k][j-2][i];
_t_50_ -= 0.2 * _t_52_;
_t_53_ = cons_4[k][j+3][i] * q_2[k][j+3][i];
_t_53_ -= cons_4[k][j-3][i] * q_2[k][j-3][i];
_t_53_ += q_4[k][j+3][i] * q_2[k][j+3][i];
_t_53_ -= q_4[k][j-3][i] * q_2[k][j-3][i];
_t_50_ += 0.038 * _t_53_;
_t_54_ = cons_4[k][j+4][i] * q_2[k][j+4][i];
_t_54_ -= cons_4[k][j-4][i] * q_2[k][j-4][i];
_t_54_ += q_4[k][j+4][i] * q_2[k][j+4][i];
_t_54_ -= q_4[k][j-4][i] * q_2[k][j-4][i];
_t_50_ -= 0.0035 * _t_54_;
_t_49_ = _t_50_ * dxinv1;
flux_4kc0jc0ic0 -= _t_49_;
flux_4[k][j][i] = flux_4kc0jc0ic0;

		flux_0[k][j][i] -= ((0.8f*(cons_3[k+1][j][i] - cons_3[k-1][j][i]) - 0.2f*(cons_3[k+2][j][i] - cons_3[k-2][j][i]) + 0.038f*(cons_3[k+3][j][i] - cons_3[k-3][j][i]) - 0.0035f*(cons_3[k+4][j][i] - cons_3[k-4][j][i]))*dxinv2); 
		flux_1[k][j][i] -= (0.8f*(cons_1[k+1][j][i]*q_3[k+1][j][i]-cons_1[k-1][j][i]*q_3[k-1][j][i])-0.2f*(cons_1[k+2][j][i]*q_3[k+2][j][i]-cons_1[k-2][j][i]*q_3[k-2][j][i])+0.038f*(cons_1[k+3][j][i]*q_3[k+3][j][i]-cons_1[k-3][j][i]*q_3[k-3][j][i])-0.0035f*(cons_1[k+4][j][i]*q_3[k+4][j][i]-cons_1[k-4][j][i]*q_3[k-4][j][i]))*dxinv2; 
		flux_2[k][j][i] -= (0.8f*(cons_2[k+1][j][i]*q_3[k+1][j][i]-cons_2[k-1][j][i]*q_3[k-1][j][i])-0.2f*(cons_2[k+2][j][i]*q_3[k+2][j][i]-cons_2[k-2][j][i]*q_3[k-2][j][i])+0.038f*(cons_2[k+3][j][i]*q_3[k+3][j][i]-cons_2[k-3][j][i]*q_3[k-3][j][i])-0.0035f*(cons_2[k+4][j][i]*q_3[k+4][j][i]-cons_2[k-4][j][i]*q_3[k-4][j][i]))*dxinv2; 
		flux_3[k][j][i] -= (0.8f*(cons_3[k+1][j][i]*q_3[k+1][j][i]-cons_3[k-1][j][i]*q_3[k-1][j][i]+(q_4[k+1][j][i]-q_4[k-1][j][i]))-0.2f*(cons_3[k+2][j][i]*q_3[k+2][j][i]-cons_3[k-2][j][i]*q_3[k-2][j][i]+(q_4[k+2][j][i]-q_4[k-2][j][i]))+0.038f*(cons_3[k+3][j][i]*q_3[k+3][j][i]-cons_3[k-3][j][i]*q_3[k-3][j][i]+(q_4[k+3][j][i]-q_4[k-3][j][i]))-0.0035f*(cons_3[k+4][j][i]*q_3[k+4][j][i]-cons_3[k-4][j][i]*q_3[k-4][j][i]+(q_4[k+4][j][i]-q_4[k-4][j][i])))*dxinv2; 
		flux_4[k][j][i] -= (0.8f*(cons_4[k+1][j][i]*q_3[k+1][j][i]-cons_4[k-1][j][i]*q_3[k-1][j][i]+(q_4[k+1][j][i]*q_3[k+1][j][i]-q_4[k-1][j][i]*q_3[k-1][j][i]))-0.2f*(cons_4[k+2][j][i]*q_3[k+2][j][i]-cons_4[k-2][j][i]*q_3[k-2][j][i]+(q_4[k+2][j][i]*q_3[k+2][j][i]-q_4[k-2][j][i]*q_3[k-2][j][i]))+0.038f*(cons_4[k+3][j][i]*q_3[k+3][j][i]-cons_4[k-3][j][i]*q_3[k-3][j][i]+(q_4[k+3][j][i]*q_3[k+3][j][i]-q_4[k-3][j][i]*q_3[k-3][j][i]))-0.0035f*(cons_4[k+4][j][i]*q_3[k+4][j][i]-cons_4[k-4][j][i]*q_3[k-4][j][i]+(q_4[k+4][j][i]*q_3[k+4][j][i]-q_4[k-4][j][i]*q_3[k-4][j][i])))*dxinv2; 
	} 
}

extern "C" void host_code (double *h_flux_0, double *h_flux_1, double *h_flux_2, double *h_flux_3, double *h_flux_4, double *h_cons_1, double *h_cons_2, double *h_cons_3, double *h_cons_4, double *h_q_1, double *h_q_2, double *h_q_3, double *h_q_4, double dxinv0, double dxinv1, double dxinv2, int L, int M, int N) {
	double *flux_0;
	hipMalloc (&flux_0, sizeof(double)*L*M*N);
	check_error ("Failed to allocate device memory for flux_0\n");
	hipMemcpy (flux_0, h_flux_0, sizeof(double)*L*M*N, hipMemcpyHostToDevice);
	double *flux_1;
	hipMalloc (&flux_1, sizeof(double)*L*M*N);
	check_error ("Failed to allocate device memory for flux_1\n");
	hipMemcpy (flux_1, h_flux_1, sizeof(double)*L*M*N, hipMemcpyHostToDevice);
	double *flux_2;
	hipMalloc (&flux_2, sizeof(double)*L*M*N);
	check_error ("Failed to allocate device memory for flux_2\n");
	hipMemcpy (flux_2, h_flux_2, sizeof(double)*L*M*N, hipMemcpyHostToDevice);
	double *flux_3;
	hipMalloc (&flux_3, sizeof(double)*L*M*N);
	check_error ("Failed to allocate device memory for flux_3\n");
	hipMemcpy (flux_3, h_flux_3, sizeof(double)*L*M*N, hipMemcpyHostToDevice);
	double *flux_4;
	hipMalloc (&flux_4, sizeof(double)*L*M*N);
	check_error ("Failed to allocate device memory for flux_4\n");
	hipMemcpy (flux_4, h_flux_4, sizeof(double)*L*M*N, hipMemcpyHostToDevice);
	double *cons_1;
	hipMalloc (&cons_1, sizeof(double)*L*M*N);
	check_error ("Failed to allocate device memory for cons_1\n");
	hipMemcpy (cons_1, h_cons_1, sizeof(double)*L*M*N, hipMemcpyHostToDevice);
	double *cons_2;
	hipMalloc (&cons_2, sizeof(double)*L*M*N);
	check_error ("Failed to allocate device memory for cons_2\n");
	hipMemcpy (cons_2, h_cons_2, sizeof(double)*L*M*N, hipMemcpyHostToDevice);
	double *cons_3;
	hipMalloc (&cons_3, sizeof(double)*L*M*N);
	check_error ("Failed to allocate device memory for cons_3\n");
	hipMemcpy (cons_3, h_cons_3, sizeof(double)*L*M*N, hipMemcpyHostToDevice);
	double *cons_4;
	hipMalloc (&cons_4, sizeof(double)*L*M*N);
	check_error ("Failed to allocate device memory for cons_4\n");
	hipMemcpy (cons_4, h_cons_4, sizeof(double)*L*M*N, hipMemcpyHostToDevice);
	double *q_1;
	hipMalloc (&q_1, sizeof(double)*L*M*N);
	check_error ("Failed to allocate device memory for q_1\n");
	hipMemcpy (q_1, h_q_1, sizeof(double)*L*M*N, hipMemcpyHostToDevice);
	double *q_2;
	hipMalloc (&q_2, sizeof(double)*L*M*N);
	check_error ("Failed to allocate device memory for q_2\n");
	hipMemcpy (q_2, h_q_2, sizeof(double)*L*M*N, hipMemcpyHostToDevice);
	double *q_3;
	hipMalloc (&q_3, sizeof(double)*L*M*N);
	check_error ("Failed to allocate device memory for q_3\n");
	hipMemcpy (q_3, h_q_3, sizeof(double)*L*M*N, hipMemcpyHostToDevice);
	double *q_4;
	hipMalloc (&q_4, sizeof(double)*L*M*N);
	check_error ("Failed to allocate device memory for q_4\n");
	hipMemcpy (q_4, h_q_4, sizeof(double)*L*M*N, hipMemcpyHostToDevice);

	dim3 blockconfig (16, 4, 4);
	dim3 gridconfig (ceil(N, blockconfig.x), ceil(M, blockconfig.y), ceil(L, blockconfig.z));
	hypterm <<<gridconfig, blockconfig>>> (flux_0, flux_1, flux_2, flux_3, flux_4, cons_1, cons_2, cons_3, cons_4, q_1, q_2, q_3, q_4, -dxinv0, dxinv1, dxinv2, L, M, N);

	hipMemcpy (h_flux_0, flux_0, sizeof(double)*L*M*N, hipMemcpyDeviceToHost);
	hipMemcpy (h_flux_1, flux_1, sizeof(double)*L*M*N, hipMemcpyDeviceToHost);
	hipMemcpy (h_flux_3, flux_3, sizeof(double)*L*M*N, hipMemcpyDeviceToHost);
	hipMemcpy (h_flux_4, flux_4, sizeof(double)*L*M*N, hipMemcpyDeviceToHost);
	hipMemcpy (h_flux_2, flux_2, sizeof(double)*L*M*N, hipMemcpyDeviceToHost);
}